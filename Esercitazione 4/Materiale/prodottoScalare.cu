#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

// Funzione per la gestione degli errori CUDA
void checkCuda(hipError_t result, const char *fn, const char *file, int line) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error at %s:%d code=%d(%s) \"%s\"\n",
                file, line, (unsigned int)result, hipGetErrorString(result), fn);
        exit(EXIT_FAILURE);
    }
}

#define CUDA_CHECK(val) checkCuda((val), #val, __FILE__, __LINE__)

void prodottoCPU(float *a, float *b, float *c, int n);
__global__ void prodottoGPU(float* a, float* b, float* c, int n);

int main(void) {
	float *a_h, *b_h, *c_h, *c_d_on_h; // host data
	float *a_d, *b_d, *c_d; // device data
	int N, nBytes;
	dim3 gridDim, blockDim;
	float elapsed_gpu, elapsed_cpu;
	hipEvent_t start_gpu, stop_gpu, start_cpu, stop_cpu;

	printf("Prodotto scalare di due vettori\n");
	printf("===============================\n");
	printf("Inserisci il numero degli elementi dei vettori: ");
	printf("\n");
	scanf("%d", &N);
	printf("Inserisci il numero di thread per blocco: ");
	scanf("%d", &blockDim.x);
	printf("\n");

	// Determinazione esatta del numero di blocchi
	gridDim = N / blockDim.x + ((N % blockDim.x) == 0 ? 0:1);

	nBytes = N * sizeof(float);
	a_h = (float *)malloc(nBytes);
	b_h = (float *)malloc(nBytes);
	c_h = (float *)malloc(nBytes);
	hipMalloc((void **) &a_d, nBytes);
	hipMalloc((void **) &b_d, nBytes);
	hipMalloc((void **) &c_d, nBytes);
	c_d_on_h = (float *)malloc(nBytes);

	// Generazione casuale inizializzata mediante il tempo corrente
	srand((unsigned int) time(0));
	for (int i = 0; i < N; i++) {
		a_h[i] = rand() % 5 - 2;
		b_h[i] = rand() % 5 - 2;;
	}

	hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, nBytes, hipMemcpyHostToDevice);

	// Azzeriamo il contenuto del vettore c
	memset(c_h, 0, nBytes);
	hipMemset(c_d, 0, nBytes);

	// Invocazione del kernel
	hipEventCreate(&start_gpu);
	hipEventCreate(&stop_gpu);
	hipEventRecord(start_gpu);
	printf("GridDim = %d, BlockDim = %d\n", gridDim.x, blockDim.x);
	prodottoGPU<<<gridDim, blockDim>>>(a_d, b_d, c_d, N);
	CUDA_CHECK(hipGetLastError());
	hipMemcpy(c_h, c_d, nBytes, hipMemcpyDeviceToHost);
	float sommaGPU = 0;
	for(int i = 0; i < N; i++){
		sommaGPU += c_h[i];
	}
	hipEventRecord(stop_gpu);
	hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&elapsed_gpu, start_gpu, stop_gpu);
	hipEventDestroy(start_gpu);
	hipEventDestroy(stop_gpu);

	hipEventCreate(&start_cpu);
	hipEventCreate(&stop_cpu);
	hipEventRecord(start_cpu);
	// Calcolo somma seriale su CPU
	prodottoCPU(a_h, b_h, c_d_on_h, N);
	float sommaCPU = 0;
	for(int i = 0; i < N; i++){
		sommaCPU += c_d_on_h[i];
	}
	hipEventRecord(stop_cpu);
	hipEventSynchronize(stop_cpu);

	hipEventElapsedTime(&elapsed_cpu, start_cpu, stop_cpu);
	hipEventDestroy(start_cpu);
	hipEventDestroy(stop_cpu);

	// Verifica che i risultati di CPU e GPU siano uguali
	// Se non stampa nulla, i due vettori sono uguali
	for (int i = 0; i <  N; i++) {
		assert( c_h[i] == c_d_on_h[i] );
	}

	if (N<20){
		for(int i = 0; i < N; i++)
			printf("a_h[%d]=%6.2f ",i, a_h[i]);
		printf("\n");
		for(int i = 0; i < N; i++)
			printf("b_h[%d]=%6.2f ",i, b_h[i]);
		printf("\n");
		for(int i = 0; i < N; i++)
			printf("c_h[%d]=%6.2f ",i, c_h[i]);
		printf("\n");
	}

	printf("Somma GPU = %f\n", sommaGPU);
	printf("Somma CPU = %f\n", sommaCPU);
	printf("time_GPU = %6.2f\n", elapsed_gpu);
	printf("time_CPU = %6.2f\n", elapsed_cpu);
	assert(sommaGPU == sommaCPU);

	free(a_h);
	free(b_h);
	free(c_h);
	free(c_d_on_h);
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
	return 0;
}

// Host
void prodottoCPU(float *a, float *b, float *c, int n) {
	for(int i = 0; i < n; i++) {
		c[i] = a[i] * b[i];
	}
}

// Device
__global__ void prodottoGPU(float* a, float * b, float* c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < n)
		c[index] = a[index] * b[index];
}
